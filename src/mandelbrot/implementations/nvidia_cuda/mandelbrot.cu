
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__device__ unsigned int mandelbrot(
    const double x,
    const double y,
    const unsigned int cutoff
) {
    hipDoubleComplex z = make_hipDoubleComplex(0.0, 0.0);
    const hipDoubleComplex c = make_hipDoubleComplex(x, y);
    int iterations = 0;
    while (iterations < cutoff && hipCabs(z) < 2.0) {
        z = hipCadd(hipCmul(z, z), c);
        ++iterations;
    }
    return iterations - 1;
}

__global__ void compute_mandelbrot(
    const int width,
    const int height,
    const double x_0, const double x_1,
    const double y_0, const double y_1,
    const unsigned int cutoff,

    unsigned int *divergence,
    const size_t N
) {
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const double x_scale = abs(x_0 - x_1) / static_cast<double>(width);
    const double y_scale = abs(y_0 - y_1) / static_cast<double>(height);

    const unsigned int _i = tid % width;
    const unsigned int _j = tid / width;
    divergence[tid] = mandelbrot(
        x_0 + static_cast<double>(_i) * x_scale,
        y_0 + static_cast<double>(_j) * y_scale,
        cutoff
    );
}


__global__ void apply_colormap(
    const unsigned int *divergence,
    const unsigned int cutoff,
    const unsigned char *colormap,
    const size_t N_colormap,
    unsigned char *pixels,
    const size_t N
) {
    const unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const unsigned char color_index = (divergence[tid] * N_colormap / cutoff);

    pixels[tid + 0 * N] = colormap[color_index + 0 * N_colormap];
    pixels[tid + 1 * N] = colormap[color_index + 1 * N_colormap];
    pixels[tid + 2 * N] = colormap[color_index + 2 * N_colormap];
}
